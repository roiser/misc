#include "dev_array.h"
#include "hst_matrix.h"
#include "kernel.h"

/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/

int main() {
  const int M = 8, N = 8, K = 4, SA = M * K, SB = K * N, SC = M * N;
  double _A_mat_[SA], _B_mat_[SB], C_rm[SC];
  dev_array<double> d_A(SA), d_B(SB), d_C(SC);

  fill(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N);

  d_A.set(_A_mat_, SA);
  d_B.set(_B_mat_, SB);

  mult<M, N, K><<<1, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C_rm, SC);
  hipDeviceSynchronize();

  print(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N);

  return 0;
}
