#include "hip/hip_runtime.h"

#include "dev_array.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <mma.h>
#include <stdlib.h>

using namespace nvcuda;

constexpr int M = 8, N = 8, K = 4;

__global__ void mult(const double *A, const double *B, double *C) {

  // printf("kernel start\n");

  wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a_frag;
  // row_major or col_major
  wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, M, N, K, double> c_frag;

  wmma::load_matrix_sync(a_frag, A, K);
  wmma::load_matrix_sync(b_frag, B, K); // row-major: M, col-major: K
  wmma::fill_fragment(c_frag, 0.);

  wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);

  wmma::store_matrix_sync(C, c_frag, N, wmma::mem_row_major);

  // printf("kernel stop\n");
}

int main() {

  std::cout << "start" << std::endl;

  const int SA = M * K, SB = K * N, SC = M * N;

  // clang-format off
  double A[SA] = {1., 2., 3., 4.,
                  1., 2., 3., 4.,
                  1., 2., 3., 4.,
                  1., 2., 3., 4.,
                  1., 2., 3., 4.,
                  1., 2., 3., 4.,
                  1., 2., 3., 4.,
                  1., 2., 3., 4.};
  // row-major
  // double B[SB] = {8., 7., 6., 5., 4., 3., 2., 1.,
  //                 8., 7., 6., 5., 4., 3., 2., 1.,
  //                 8., 7., 6., 5., 4., 3., 2., 1.,
  //                 8., 7., 6., 5., 4., 3., 2., 1.};
  // col-major
  double B[SB] = {8., 8., 8., 8.,
                  7., 7., 7., 7.,
                  6., 6., 6., 6.,
                  5., 5., 5., 5.,
                  4., 4., 4., 4.,
                  3., 3., 3., 3.,
                  2., 2., 2., 2.,
                  1., 1., 1., 1.};
  double C[SC] = {0., 0., 0., 0., 0., 0., 0., 0.,
                  0., 0., 0., 0., 0., 0., 0., 0.,
                  0., 0., 0., 0., 0., 0., 0., 0.,
                  0., 0., 0., 0., 0., 0., 0., 0.,
                  0., 0., 0., 0., 0., 0., 0., 0.,
                  0., 0., 0., 0., 0., 0., 0., 0.,
                  0., 0., 0., 0., 0., 0., 0., 0.,
                  0., 0., 0., 0., 0., 0., 0., 0.};
// clang-format once

  dev_array<double> d_A(SA);
  dev_array<double> d_B(SB);
  dev_array<double> d_C(SC);

  d_A.set(A, SA);
  d_B.set(B, SB);

  mult<<<1, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C, SC);
  hipDeviceSynchronize();

  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      std::cout << C[i * M + j] << ", ";
    }
    std::cout << std::endl;
  }

  std::cout << "stop" << std::endl;

  return 0;
}
