#include "hip/hip_runtime.h"
#include "dev_array.h"
#include "hst_matrix.h"
#include "kernel.h"

#include <iostream>
#include <hip/hip_complex.h>
#include <hipblas.h>


/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/


int mult(hipblasHandle_t handle, const double *d_A, const double *d_B, double *d_C, double *d_y, double *h_y, int dsize) {

  hipblasStatus_t cublas_status;
  hipError_t cuda_status;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = HIPBLAS_OP_N;

  int m = 24, n = 1, lda = 24, ldb = 24, ldc = 24;
  double alpha = 1, beta = 0;

  cublas_status = hipblasDsymm(handle, side, uplo, m, n, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);

  int incx = 1, incy = 1;
  m = 1;
  n = 24;
  lda = 1;

  cublas_status = hipblasDgemv(handle, trans, m, n, &alpha, d_B, lda, d_C, incx, &beta, d_y, incy);
  cuda_status = hipMemcpy(h_y, d_y, dsize, hipMemcpyDeviceToHost);

  return max(cublas_status, cuda_status);;

}

int main() {

  hipblasHandle_t handle;
  hipError_t cuda_status;

  int dsize = sizeof(double),
      vsize = dsize * medim,
      msize = vsize * medim,
      mult_status = 0;
  const double
    *h_A = (double *)malloc(msize),
    *h_B = (double *)malloc(vsize),
    *d_A, *d_B;
  double
    *h_C = (double *)malloc(vsize), 
    *h_y = (double*) malloc(dsize),
    *d_C, *d_y;

  cuda_status = hipMalloc((void**) &d_A, msize);
  cuda_status = hipMalloc((void**) &d_B, vsize);
  cuda_status = hipMalloc((void**) &d_C, vsize);
  cuda_status = hipMalloc((void**) &d_y, dsize);

  memcpy((void*)h_A, &cf[0], msize);
  cuda_status = hipMemcpy((void*)d_A, h_A, msize, hipMemcpyHostToDevice);

  hipblasCreate(&handle);

  memcpy((void*)h_B, &jamp0r[0], vsize);
  cuda_status = hipMemcpy((void*)d_B, h_B, vsize, hipMemcpyHostToDevice);

  mult_status = mult(handle, d_A, d_B, d_C, d_y, h_y, dsize);

  std::cout << "y: " << *h_y << std::endl;

  hipblasDestroy(handle);

  return max(mult_status, cuda_status);

}


// https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemv

// alpha*A*B + beta*C (side=left) or alpha*B*A + beta*C (side=right),  A is symmetric
// hipblasHandle_t handle,    // 
// hipblasSideMode_t side     // HIPBLAS_SIDE_LEFT or HIPBLAS_SIDE_RIGHT (A is on the left or right side)
// hipblasFillMode_t uplo,    // HIPBLAS_FILL_MODE_LOWER (0) or HIPBLAS_FILL_MODE_UPPER (1), lower or upper part is referenced
// int m, int n,             // number of rows (m) or cols (n) of matrix C and B, with matrix A sized accordingly. 
// const double *alpha,      // <type> scalar used for multiplication
// const double *A,          // <type> array of dimension lda x m with lda>=max(1,m) if side == HIPBLAS_SIDE_LEFT and lda x n with lda>=max(1,n) otherwise.
// const double *B,          // <type> array of dimension ldb x n with ldb>=max(1,m). 
// const double *beta,       // <type> scalar used for multiplication, if beta == 0 then C does not have to be a valid input.
// double *C                 // <type> array of dimension ldb x n with ldb>=max(1,m).
// int lda, ldb, ldc         // leading dimension of two-dimensional array used to store matrix A or B or C

// hipblasStatus_t hipblasDsymm(hipblasHandle_t handle,
//                            hipblasSideMode_t side, hipblasFillMode_t uplo,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *B, int ldb,
//                            const double          *beta,
//                            double          *C, int ldc)



// alpha*A(x) + beta*y
// hipblasOperation_t trans,      // operation op(A) that is non- or (conj.) transpose. HIPBLAS_OP_N/T/H
// int m, int n,                 // number of rows/cols of A
// const double *x,              // vector x
// double *y,                    // vector y
// int incx, incy                // stride between consecutive elements of x/y. 


// hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *x, int incx,
//                            const double          *beta,
//                            double          *y, int incy)

