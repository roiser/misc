#include "hip/hip_runtime.h"
//#define DOUBLEPRECISION
//#define CONJUGATE

#ifdef DOUBLEPRECISION
#define TTYPE double
#define CUB_SYMV hipblasDsymm
#define CUB_GEMV hipblasDgemv
// #define CUB_GEMV hipblasDgemvBatched // hipblasDgemv
#else
#define TTYPE float
#define CUB_SYMV hipblasSsymm
#define CUB_GEMV hipblasSgemv
// #define CUB_GEMV hipblasSgemvBatched // hipblasSgemv
#endif

#include "data.h"
#include "timer.h"

using namespace mgOnGpu;

#include <complex>
#include <hipblas.h>
#include <iostream>

/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/


//
// org implementation on host
//
TTYPE mult_native_host(TTYPE *cf, std::complex<TTYPE> *jamp) {
  int ncolor = 24;
  TTYPE deltaME = 0;
  for (int icol = 0; icol < ncolor; icol++) {
    std::complex<TTYPE> ztemp;
    for (int jcol = 0; jcol < ncolor; jcol++) {
      ztemp += cf[icol * ncolor + jcol] * jamp[jcol];
    }
    deltaME += (ztemp.real() * jamp[icol].real() +
                ztemp.imag() * jamp[icol].imag()); // / denom[icol];
  }
  return deltaME;
}


//
// org implementation on device
//
__global__ void mult_native_device(const TTYPE *cf, const TTYPE *jampr,
                                   const TTYPE *jampi, TTYPE *deltaME) {
  int ncolor = 24;
  *deltaME = 0;
  for (int icol = 0; icol < ncolor; icol++) {
    TTYPE ztempr = 0, ztempi = 0;
    for (int jcol = 0; jcol < ncolor; jcol++) {
      ztempr += cf[icol * ncolor + jcol] * jampr[jcol];
      ztempi += cf[icol * ncolor + jcol] * jampi[jcol];
    }
    *deltaME += (ztempr * jampr[icol] + ztempi * jampi[icol]); // / denom[icol];
  }
}


//
// cublas implementation
//
int mult_cublas(hipblasHandle_t handle, const TTYPE *d_A, const TTYPE *d_B,
                TTYPE *d_C, TTYPE *d_y, TTYPE *h_y, int dsize, float &time,
                int nevt) {

  hipblasStatus_t cubstat;
  hipError_t cudstat;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = HIPBLAS_OP_N;

  Timer<std::chrono::high_resolution_clock> t;
  int ncol = 24, incx = 1, incy = 1;
  TTYPE alpha = 1, beta = 0;

  t.Start();
  cubstat = CUB_SYMV(handle, side, uplo, ncol, nevt, &alpha, d_A, ncol, d_B, ncol, &beta, d_C, ncol);
  cubstat = CUB_GEMV(handle, trans, nevt, ncol, &alpha, d_B, nevt, d_C, incx, &beta, d_y, incy);
  // cubstat = CUB_GEMV(handle, trans, 1, ncol, &alpha, d_B, nevt, d_C, ncol, &beta, d_y, ncol, nevt);
  time += t.GetDuration();

  cudstat = hipMemcpy(h_y, d_y, dsize, hipMemcpyDeviceToHost);

  return max(cubstat, cudstat);
}


//
// main
//
int main() {

  int nevt = 1;

  hipblasHandle_t handle;
  hipError_t cuda_status;

  Timer<std::chrono::high_resolution_clock> t;
  float time = 0.;

  int dsize = sizeof(TTYPE), vsize = dsize * medim, msize = vsize * medim,
      mult_status = 0;
  const TTYPE *h_A = (TTYPE *)malloc(msize), // color matrix
      *h_B = (TTYPE *)malloc(vsize * nevt),  // jamps
      *d_A, *d_Br, *d_Bi, *tmp;
  TTYPE *h_C = (TTYPE *)malloc(vsize * nevt), // temp result
      *h_y = (TTYPE *)malloc(dsize * nevt),   // matrix elements
      *d_C, *d_y, me = 0, me2 = 0;

  cuda_status = hipMalloc((void **)&d_A, msize);  // color matrix
  cuda_status = hipMalloc((void **)&d_Br, vsize * nevt); // jamps real
  cuda_status = hipMalloc((void **)&d_Bi, vsize * nevt); // ramps imag
  cuda_status = hipMalloc((void **)&d_C, vsize * nevt);  // temp result
  cuda_status = hipMalloc((void **)&d_y, dsize * nevt);  // matrix elements


  //
  // prepare memory
  //
  memcpy((void *)h_A, &cf[0], msize);
  cuda_status = hipMemcpy((void *)d_A, h_A, msize, hipMemcpyHostToDevice);

  tmp = h_B;
  for (int i = 0; i < nevt; ++i) {
    memcpy((void *)tmp, &jamp0r[0], vsize);
    tmp += 24;
  }
  cuda_status = hipMemcpy((void *)d_Br, h_B, vsize * nevt, hipMemcpyHostToDevice);

  tmp = h_B;
  for (int i = 0; i < nevt; ++i) {
    memcpy((void *)tmp, &jamp0i[0], vsize);
    tmp += 24;
  }
  cuda_status = hipMemcpy((void *)d_Bi, h_B, vsize * nevt, hipMemcpyHostToDevice);


  //
  // conjugate if needed
  //
#ifdef CONJUGATE
  for (int i = 0; i < medim * nevt; ++i) h_Bi[i] = -1 * h_Bi[i];
  cuda_status = hipMemcpy((void *)d_Bi, h_Bi, vsize * nevt, hipMemcpyHostToDevice);
#endif

  //
  // cublas
  //
  hipblasCreate(&handle);
  mult_status = mult_cublas(handle, d_A, d_Br, d_C, d_y, h_y, dsize, time, nevt);
  me += *h_y;
  mult_status = mult_cublas(handle, d_A, d_Bi, d_C, d_y, h_y, dsize, time, nevt);
  me += *h_y;
  hipblasDestroy(handle);
  std::cout << "cublas    : " << me << ", " << time << std::endl;

  //
  // org on host
  //
  std::complex<TTYPE> jamp[vsize];
  for (int i = 0; i < vsize; ++i) {
    jamp[i] = std::complex<TTYPE>(jamp0r[i], jamp0i[i]);
  }
  time = 0.;
  t.Start();
  me2 = mult_native_host(cf, jamp);
  std::cout << "org host  : " << me2 << ", " << t.GetDuration() << std::endl;

  //
  // org on device
  // 
  time = 0.;
  t.Start();
  mult_native_device<<<1, 1>>>(d_A, d_Br, d_Bi, d_y);
  cuda_status = hipMemcpy(h_y, d_y, dsize, hipMemcpyDeviceToHost);
  std::cout << "org device: " << *h_y << ", " << t.GetDuration() << std::endl;

  return max(mult_status, cuda_status);
}

// https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemv

// alpha*A*B + beta*C (side=left) or alpha*B*A + beta*C (side=right),  A is
// symmetric hipblasHandle_t handle,    // hipblasSideMode_t side     //
// HIPBLAS_SIDE_LEFT or HIPBLAS_SIDE_RIGHT (A is on the left or right side)
// hipblasFillMode_t uplo,    // HIPBLAS_FILL_MODE_LOWER (0) or
// HIPBLAS_FILL_MODE_UPPER (1), lower or upper part is referenced int m, int n,
// // number of rows (m) or cols (n) of matrix C and B, with matrix A sized
// accordingly. const double *alpha,      // <type> scalar used for
// multiplication const double *A,          // <type> array of dimension lda x m
// with lda>=max(1,m) if side == HIPBLAS_SIDE_LEFT and lda x n with lda>=max(1,n)
// otherwise. const double *B,          // <type> array of dimension ldb x n
// with ldb>=max(1,m). const double *beta,       // <type> scalar used for
// multiplication, if beta == 0 then C does not have to be a valid input. double
// *C                 // <type> array of dimension ldb x n with ldb>=max(1,m).
// int lda, ldb, ldc         // leading dimension of two-dimensional array used
// to store matrix A or B or C

// hipblasStatus_t hipblasDsymm(hipblasHandle_t handle,
//                            hipblasSideMode_t side, hipblasFillMode_t uplo,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *B, int ldb,
//                            const double          *beta,
//                            double          *C, int ldc)

// alpha*A(x) + beta*y
// hipblasOperation_t trans,      // operation op(A) that is non- or (conj.)
// transpose. HIPBLAS_OP_N/T/H int m, int n,                 // number of
// rows/cols of A const double *x,              // vector x double *y, // vector
// y int incx, incy                // stride between consecutive elements of
// x/y.

// hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *x, int incx,
//                            const double          *beta,
//                            double          *y, int incy)
