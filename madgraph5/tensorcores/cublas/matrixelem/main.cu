#include "hip/hip_runtime.h"
#define DOUBLEPRECISION
#define USE_NVTX

#ifdef USE_NVTX
#include "nvToolsExt.h"

const uint32_t colors[] = {0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff,
                           0xff00ffff, 0xffff0000, 0xffffffff, 0x00ffffff};
const int num_colors = sizeof(colors) / sizeof(uint32_t);

#define PUSH_RANGE(name, cid)                                                  \
  {                                                                            \
    int color_id = cid;                                                        \
    color_id = color_id % num_colors;                                          \
    nvtxEventAttributes_t eventAttrib = {0};                                   \
    eventAttrib.version = NVTX_VERSION;                                        \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;                          \
    eventAttrib.colorType = NVTX_COLOR_ARGB;                                   \
    eventAttrib.color = colors[color_id];                                      \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII;                         \
    eventAttrib.message.ascii = name;                                          \
    nvtxRangePushEx(&eventAttrib);                                             \
  }
#define POP_RANGE nvtxRangePop();
#else
#define PUSH_RANGE(name, cid)
#define POP_RANGE
#endif

#if defined(DOUBLEPRECISION)
#define TTYPE double
#define CUB_SYMV hipblasDsymm
#define CUB_GEMV hipblasDgemvBatched
#else // DOUBLEPRECISION
#define TTYPE float
#define CUB_SYMV hipblasSsymm
#define CUB_GEMV hipblasSgemvBatched
#endif // DOUBLEPRECISION

#define cudaCheckError()                                                       \
  {                                                                            \
    hipError_t e = hipGetLastError();                                        \
    if (e != hipSuccess) {                                                    \
      printf("Cuda error %s:%d: '%s'\n", __FILE__, __LINE__,                   \
             hipGetErrorString(e));                                           \
      exit(0);                                                                 \
    }                                                                          \
  }

#include "data.h"
#include "timer.h"

using namespace mgOnGpu;

#include <complex>
#include <hipblas.h>
#include <iostream>

//
// org implementation on host
//
TTYPE mult_native_host(TTYPE *cf, std::complex<TTYPE> *jamp, int nevt) {
  int ncolor = 24;
  TTYPE deltaME = 0;
  for (int i = 0; i < nevt; ++i) {
    deltaME = 0;
    for (int icol = 0; icol < ncolor; icol++) {
      std::complex<TTYPE> ztemp;
      for (int jcol = 0; jcol < ncolor; jcol++) {
        ztemp += cf[icol * ncolor + jcol] * jamp[jcol];
      }
      deltaME += (ztemp.real() * jamp[icol].real() +
                  ztemp.imag() * jamp[icol].imag()); // / denom[icol];
    }
  }
  return deltaME;
}

//
// org implementation on device
//
__global__ void mult_native_device(const TTYPE *cf, const TTYPE *jampr,
                                   const TTYPE *jampi, TTYPE *deltaME,
                                   int ncol) {
  *deltaME = 0;
  for (int icol = 0; icol < ncol; icol++) {
    TTYPE ztempr = 0, ztempi = 0;
    for (int jcol = 0; jcol < ncol; jcol++) {
      ztempr += cf[icol * ncol + jcol] * jampr[jcol];
      ztempi += cf[icol * ncol + jcol] * jampi[jcol];
    }
    *deltaME += (ztempr * jampr[icol] + ztempi * jampi[icol]); // / denom[icol];
  }
}

//
// kernel to set the pointers to arrays
//
__global__ void setMem(const TTYPE *d_Br, const TTYPE *d_Bi, TTYPE *d_C,
                       TTYPE *d_y, const TTYPE **d_BBr, const TTYPE **d_BBi,
                       TTYPE **d_CC, TTYPE **d_yy, int ncol, int nevt) {
  for (int i = 0; i < nevt; ++i) {
    d_BBr[i] = &d_Br[i * ncol];
    d_BBi[i] = &d_Bi[i * ncol];
    d_CC[i] = &d_C[i * ncol];
    d_yy[i] = &d_y[i];
  }
}

//
// cublas implementation
//
void mult_cublas(hipblasHandle_t handle, const TTYPE *d_A, const TTYPE *d_B,
                 TTYPE *d_C, TTYPE *d_y, const TTYPE *d_BB, TTYPE *d_CC,
                 TTYPE *d_yy, int dsize, float &time, int ncol, int nevt) {

  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t transn = HIPBLAS_OP_N;

  Timer<std::chrono::high_resolution_clock> t;
  TTYPE alpha = 1, beta = 0;

  t.Start();

  PUSH_RANGE("5 - cublas symv", 5)
  CUB_SYMV(handle, side, uplo, ncol, nevt, &alpha, d_A, ncol, d_B, ncol, &beta,
           d_C, ncol);
  POP_RANGE
  cudaCheckError();

  PUSH_RANGE("6 - cublas gemv", 6)
  CUB_GEMV(handle, transn, 1, ncol, &alpha, (const TTYPE **)d_BB, 1,
           (const TTYPE **)d_CC, 1, &beta, (TTYPE **)d_yy, 1, nevt);
  POP_RANGE
  cudaCheckError();

  time += t.GetDuration();
}

void usage() {
  std::cout << "./main #threads/block #blocks/grid" << std::endl;
  exit(1);
}

//
// main
//
int main(int argc, char **argv) {

  if (argc != 3)
    usage();

  int threads = std::stoi(argv[1]), blocks = std::stoi(argv[2]);
  int nevt = threads * blocks, ncol = 24;

  hipblasHandle_t handle;

  Timer<std::chrono::high_resolution_clock> t;
  float time = 0.;

  int psize = sizeof(TTYPE *), dsize = sizeof(TTYPE), vsize = dsize * medim,
      msize = vsize * medim;
  const TTYPE *h_A = (TTYPE *)malloc(msize), // color matrix
      *h_B = (TTYPE *)malloc(vsize * nevt),  // jamps
      *d_A, *d_Br, *d_Bi, *d_BBr, *d_BBi, *tmp;
  TTYPE *h_C = (TTYPE *)malloc(vsize * nevt), // temp result
      *h_y = (TTYPE *)malloc(dsize * nevt),   // matrix elements
      *d_C, *d_CC, *d_y, *d_yy, me = 0, me2 = 0;
  TTYPE **h_CC = new TTYPE *[nevt](); // initialize temp result

  //
  // prepare memory
  //
  PUSH_RANGE("0 - cuda malloc memory", 0)
  hipMalloc((void **)&d_A, msize); // color matrix
  cudaCheckError();
  hipMalloc((void **)&d_Br, vsize * nevt); // jamps real
  cudaCheckError();
  hipMalloc((void **)&d_Bi, vsize * nevt); // ramps imag
  cudaCheckError();
  hipMalloc((void **)&d_C, vsize * nevt); // temp result
  cudaCheckError();
  hipMalloc((void **)&d_y, dsize * nevt); // matrix elements
  cudaCheckError();

  hipMalloc((void **)&d_BBr, psize * nevt); // batch gemv
  cudaCheckError();
  hipMalloc((void **)&d_BBi, psize * nevt); // batch gemv
  cudaCheckError();
  hipMalloc((void **)&d_CC, psize * nevt); // batch gemv
  cudaCheckError();
  hipMalloc((void **)&d_yy, psize * nevt); // batch gemv
  cudaCheckError();
  POP_RANGE

  PUSH_RANGE("1 - copy memory", 1)
  memcpy((void *)h_A, &cf[0], msize);
  hipMemcpy((void *)d_A, h_A, msize, hipMemcpyHostToDevice);
  cudaCheckError();

  tmp = h_B;
  for (int i = 0; i < nevt; ++i) {
    memcpy((void *)tmp, &jamp0r[0], vsize);
    tmp += 24;
  }
  hipMemcpy((void *)d_Br, h_B, vsize * nevt, hipMemcpyHostToDevice);
  cudaCheckError();

  tmp = h_B;
  for (int i = 0; i < nevt; ++i) {
    memcpy((void *)tmp, &jamp0i[0], vsize);
    tmp += 24;
  }
  hipMemcpy((void *)d_Bi, h_B, vsize * nevt, hipMemcpyHostToDevice);
  cudaCheckError();

  hipMemcpy((void *)d_CC, h_CC, psize * nevt, hipMemcpyHostToDevice);
  cudaCheckError();

  POP_RANGE

  //
  // cublas
  //
  PUSH_RANGE("2 - prepare cublas", 2)
  hipblasCreate(&handle);
  cudaCheckError();

  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
  cudaCheckError();

  setMem<<<1, 1>>>(d_Br, d_Bi, d_C, d_y, (const TTYPE **)d_BBr,
                   (const TTYPE **)d_BBi, (TTYPE **)d_CC, (TTYPE **)d_yy, ncol,
                   nevt);
  cudaCheckError();
  POP_RANGE

  for (int i = 0; i < 10; ++i) {
    me = 0.;
    time = 0.;
    mult_cublas(handle, d_A, d_Br, d_C, d_y, d_BBr, d_CC, d_yy, dsize, time,
                ncol, nevt);
    hipMemcpy(h_y, d_y, dsize * nevt, hipMemcpyDeviceToHost);
    cudaCheckError();
    me += h_y[0];
    mult_cublas(handle, d_A, d_Bi, d_C, d_y, d_BBi, d_CC, d_yy, dsize, time,
                ncol, nevt);
    hipMemcpy(h_y, d_y, dsize * nevt, hipMemcpyDeviceToHost);
    cudaCheckError();
    me += h_y[0];
    std::cout << "cublas    : " << me << ", " << time << std::endl;
  }

  hipblasDestroy(handle);
  cudaCheckError();

  //
  // org on host
  //
  PUSH_RANGE("3 - compute org on host", 3)
  std::complex<TTYPE> jamp[vsize];
  for (int i = 0; i < vsize; ++i) {
    jamp[i] = std::complex<TTYPE>(jamp0r[i], jamp0i[i]);
  }
  time = 0.;
  t.Start();
  me2 = mult_native_host(cf, jamp, nevt);
  std::cout << "org host  : " << me2 << ", " << t.GetDuration() << std::endl;
  POP_RANGE

  //
  // org on device
  //
  for (int i = 0; i < 10; ++i) {
    time = 0.;
    t.Start();
    PUSH_RANGE("4 - compute org on device", 4)
    mult_native_device<<<threads, blocks>>>(d_A, d_Br, d_Bi, d_y, ncol);
    POP_RANGE
    cudaCheckError();
    time = t.GetDuration();
    hipMemcpy(h_y, d_y, dsize * nevt, hipMemcpyDeviceToHost);
    cudaCheckError();
    std::cout << "org device: " << *h_y << ", " << time << std::endl;
  }

  return 0;
}
