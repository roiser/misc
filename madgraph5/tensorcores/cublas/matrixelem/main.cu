#include "hip/hip_runtime.h"
//#define DOUBLEPRECISION
//#define COMPLEXCONJUGATE

#define NEWSIGNATURE_GEMV // <-- <-- <--
//#define NEWSIGNATURE_GEMM

#if defined(DOUBLEPRECISION)
#define TTYPE double
#define CUB_SYMV hipblasDsymm

#if defined(NEWSIGNATURE_GEMV)
#define SETMEM
#define CUB_GEMV hipblasDgemvBatched
#elif defined(NEWSIGNATURE_GEMM)
#define CUB_GEMV hipblasDgemmBatched
#else
#define CUB_GEMV hipblasDgemv
#endif // NEWSIGNATURE_GEMV

#else // DOUBLEPRECISION
#define TTYPE float
#define CUB_SYMV hipblasSsymm

#if defined(NEWSIGNATURE_GEMV)
#define SETMEM
#define CUB_GEMV hipblasSgemvBatched
#elif defined(NEWSIGNATURE_GEMM)
#define CUB_GEMV hipblasSgemmBatched
#else
#define CUB_GEMV hipblasSgemv
#endif // NEWSIGNATURE_GEMV
#endif // DOUBLEPRECISION

#include "data.h"
#include "timer.h"

using namespace mgOnGpu;

#include <complex>
#include <hipblas.h>
#include <iostream>

/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/

//
// org implementation on host
//
TTYPE mult_native_host(TTYPE *cf, std::complex<TTYPE> *jamp) {
  int ncolor = 24;
  TTYPE deltaME = 0;
  for (int icol = 0; icol < ncolor; icol++) {
    std::complex<TTYPE> ztemp;
    for (int jcol = 0; jcol < ncolor; jcol++) {
      ztemp += cf[icol * ncolor + jcol] * jamp[jcol];
    }
    deltaME += (ztemp.real() * jamp[icol].real() +
                ztemp.imag() * jamp[icol].imag()); // / denom[icol];
  }
  return deltaME;
}

//
// org implementation on device
//
__global__ void mult_native_device(const TTYPE *cf, const TTYPE *jampr,
                                   const TTYPE *jampi, TTYPE *deltaME) {
  int ncolor = 24;
  *deltaME = 0;
  for (int icol = 0; icol < ncolor; icol++) {
    TTYPE ztempr = 0, ztempi = 0;
    for (int jcol = 0; jcol < ncolor; jcol++) {
      ztempr += cf[icol * ncolor + jcol] * jampr[jcol];
      ztempi += cf[icol * ncolor + jcol] * jampi[jcol];
    }
    *deltaME += (ztempr * jampr[icol] + ztempi * jampi[icol]); // / denom[icol];
  }
}

//
// kernel to set the pointers to arrays
//
__global__ void setMem(const TTYPE *d_B, TTYPE *d_C, TTYPE *d_y,
                       const TTYPE **d_BB, TTYPE **d_CC, TTYPE **d_yy, int ncol,
                       int nevt) {
  // sr war TTYPE *d_XX[nevt]
  for (int i = 0; i < nevt; ++i) {
    // d_y[i] = 0.;
    d_BB[i] = &d_B[i * ncol];
    d_CC[i] = &d_C[i * ncol];
    d_yy[i] = &d_y[i];
    // printf("%f\n", d_BB[i][0]);
  }
}

//
// print mem
//
__global__ void printMem(TTYPE *d_y, TTYPE **d_yy, int nevt) {
  for (int i = 0; i < nevt; ++i) {
    printf("kernel d_y, evt %d: %f", i, d_y[i]);
#if defined(SETMEM)
    printf(", %f", d_yy[i][0]);
#endif
    printf("\n");
  }
}

//
// cublas implementation
//
int mult_cublas(hipblasHandle_t handle, const TTYPE *d_A, const TTYPE *d_B,
                TTYPE *d_C, TTYPE *d_y, const TTYPE *d_BB, TTYPE *d_CC,
                TTYPE *d_yy, int dsize, float &time, int nevt) {

  hipblasStatus_t cubstat;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t transn = HIPBLAS_OP_N, transt = HIPBLAS_OP_T;

  Timer<std::chrono::high_resolution_clock> t;
  int ncol = 24;
  TTYPE alpha = 1, beta = 0;

  t.Start();
  // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-symm
  cubstat = CUB_SYMV(handle, side, uplo, ncol, nevt, &alpha, d_A, ncol, d_B,
                     ncol, &beta, d_C, ncol);
  hipDeviceSynchronize();

#if defined(SETMEM)
  setMem<<<1, 1>>>(d_B, d_C, d_y, (const TTYPE **)d_BB, (TTYPE **)d_CC,
                   (TTYPE **)d_yy, ncol, nevt);
  hipDeviceSynchronize();
#endif

#if defined(NEWSIGNATURE_GEMV)
  // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemvbatched
  cubstat = CUB_GEMV(handle, transn, 1, ncol, &alpha, (TTYPE **)d_BB, ncol,
                     (TTYPE **)d_CC, ncol, &beta, (TTYPE **)d_yy, 1, nevt);
  hipDeviceSynchronize();
#elif defined(NEWSIGNATURE_GEMM)
  // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemmbatched
  cubstat =
      CUB_GEMV(handle, transn, transn, 1, 1, ncol, &alpha, (TTYPE **)d_BB, ncol,
               (TTYPE **)d_CC, ncol, &beta, (TTYPE **)d_yy, 1, nevt);
#else  // NEWSIGNATURE_GEMV
  int incx = 1, incy = 1;
  cubstat = CUB_GEMV(handle, transn, nevt, ncol, &alpha, d_B, nevt, d_C, incx,
                     &beta, d_y, incy);
#endif // NEWSIGNATURE_GEMV

  time += t.GetDuration();

  printMem<<<1, 1>>>(d_y, (TTYPE **)d_yy, nevt);

  return cubstat;
}

//
// main
//
int main() {

  int nevt = 1;

  hipblasHandle_t handle;
  hipError_t custat;

  Timer<std::chrono::high_resolution_clock> t;
  float time = 0.;

  int psize = sizeof(TTYPE *), dsize = sizeof(TTYPE), vsize = dsize * medim,
      msize = vsize * medim, mult_status = 0;
  const TTYPE *h_A = (TTYPE *)malloc(msize), // color matrix
      *h_B = (TTYPE *)malloc(vsize * nevt),  // jamps
      *d_A, *d_Br, *d_Bi, *d_BB, *tmp;
  TTYPE *h_C = (TTYPE *)malloc(vsize * nevt), // temp result
      *h_y = (TTYPE *)malloc(dsize * nevt),   // matrix elements
      *d_C, *d_CC, *d_y, *d_yy, me = 0, me2 = 0;
  TTYPE **h_CC = new TTYPE *[nevt](); // initialize temp result

  //
  // prepare memory
  //
  custat = hipMalloc((void **)&d_A, msize);         // color matrix
  custat = hipMalloc((void **)&d_Br, vsize * nevt); // jamps real
  custat = hipMalloc((void **)&d_Bi, vsize * nevt); // ramps imag
  custat = hipMalloc((void **)&d_C, vsize * nevt);  // temp result
  custat = hipMalloc((void **)&d_y, dsize * nevt);  // matrix elements

  custat = hipMalloc((void **)&d_BB, psize * nevt); // batch gemv
  custat = hipMalloc((void **)&d_CC, psize * nevt); // batch gemv
  custat = hipMalloc((void **)&d_yy, psize * nevt); // batch gemv

  memcpy((void *)h_A, &cf[0], msize);
  custat = hipMemcpy((void *)d_A, h_A, msize, hipMemcpyHostToDevice);

  tmp = h_B;
  for (int i = 0; i < nevt; ++i) {
    memcpy((void *)tmp, &jamp0r[0], vsize);
    tmp += 24;
  }
  custat = hipMemcpy((void *)d_Br, h_B, vsize * nevt, hipMemcpyHostToDevice);

  // debug h_Br
  // custat = hipMemcpy(h_C, d_C, vsize * nevt, hipMemcpyDeviceToHost);
  // for (int i = 0; i < medim * nevt; ++i) {
  //   std::cout << h_B[i] << ", ";
  //   if ((i + 1) % medim == 0)
  //     std::cout << std::endl;
  // }
  // std::cout << std::endl;

  tmp = h_B;
  for (int i = 0; i < nevt; ++i) {
    memcpy((void *)tmp, &jamp0i[0], vsize);
    tmp += 24;
  }
  custat = hipMemcpy((void *)d_Bi, h_B, vsize * nevt, hipMemcpyHostToDevice);

  // debug h_Bi
  // custat = hipMemcpy(h_C, d_C, vsize * nevt, hipMemcpyDeviceToHost);
  // for (int i = 0; i < medim * nevt; ++i) {
  //   std::cout << h_B[i] << ", ";
  //   if ((i + 1) % medim == 0)
  //     std::cout << std::endl;
  // }
  // std::cout << std::endl;

  custat = hipMemcpy((void *)d_CC, h_CC, psize * nevt, hipMemcpyHostToDevice);

  //
  // conjugate if needed
  //
#ifdef COMPLEXCONJUGATE
  for (int i = 0; i < medim * nevt; ++i)
    h_Bi[i] = -1 * h_Bi[i];
  custat = hipMemcpy((void *)d_Bi, h_Bi, vsize * nevt, hipMemcpyHostToDevice);
#endif // COMPLEXCONJUGATE

  //
  // cublas
  //
  hipblasCreate(&handle);
  mult_status = mult_cublas(handle, d_A, d_Br, d_C, d_y, d_BB, d_CC, d_yy,
                            dsize, time, nevt);
  hipDeviceSynchronize();

  // debug h_C
  // custat = hipMemcpy(h_C, d_C, vsize * nevt, hipMemcpyDeviceToHost);
  // std::cout << "host   h_C: ";
  // for (int i = 0; i < medim * nevt; ++i) {
  //   std::cout << h_C[i] << ", ";
  //   if ((i + 1) % medim == 0)
  //     std::cout << std::endl;
  // }
  // std::cout << std::endl;

  custat = hipMemcpy(h_y, d_y, dsize * nevt, hipMemcpyDeviceToHost);
  me += h_y[0];
  for (int i = 0; i < nevt; ++i)
    std::cout << "host   h_y, evt " << i << ": " << h_y[i] << std::endl;
  mult_status = mult_cublas(handle, d_A, d_Bi, d_C, d_y, d_BB, d_CC, d_yy,
                            dsize, time, nevt);
  hipDeviceSynchronize();

  // debug h_C
  // custat = hipMemcpy(h_C, d_C, vsize * nevt, hipMemcpyDeviceToHost);
  // std::cout << "host   h_C: ";
  // for (int i = 0; i < medim * nevt; ++i) {
  //   std::cout << h_C[i] << ", ";
  //   if ((i + 1) % medim == 0)
  //     std::cout << std::endl;
  // }
  // std::cout << std::endl;

  custat = hipMemcpy(h_y, d_y, dsize * nevt, hipMemcpyDeviceToHost);
  me += h_y[0];
  for (int i = 0; i < nevt; ++i)
    std::cout << "host   h_y, evt " << i << ": " << h_y[i] << std::endl;
  std::cout << "cublas    : " << me << ", " << time << std::endl;
  hipblasDestroy(handle);

  //
  // org on host
  //
  std::complex<TTYPE> jamp[vsize];
  for (int i = 0; i < vsize; ++i) {
    jamp[i] = std::complex<TTYPE>(jamp0r[i], jamp0i[i]);
  }
  time = 0.;
  t.Start();
  me2 = mult_native_host(cf, jamp);
  std::cout << "org host  : " << me2 << ", " << t.GetDuration() << std::endl;

  //
  // org on device
  //
  time = 0.;
  t.Start();
  mult_native_device<<<1, 1>>>(d_A, d_Br, d_Bi, d_y);
  custat = hipMemcpy(h_y, d_y, dsize * nevt, hipMemcpyDeviceToHost);
  std::cout << "org device: " << *h_y << ", " << t.GetDuration() << std::endl;

  return max(mult_status, custat);
}

//
//
// https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemmbatched
//
// hipblasStatus_t hipblasSgemmBatched(hipblasHandle_t handle,
//                                   hipblasOperation_t transa,
//                                   hipblasOperation_t transb,
//                                   int m, int n, int k,
//                                   const float           *alpha,
//                                   const float           *Aarray[], int lda,
//                                   const float           *Barray[], int ldb,
//                                   const float           *beta,
//                                   float           *Carray[], int ldc,
//                                   int batchCount)

//
//
// https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemv
//
// alpha*A*B + beta*C (side=left) or alpha*B*A + beta*C (side=right),  A is
// symmetric
//
// hipblasHandle_t handle,    // hipblasSideMode_t side HIPBLAS_SIDE_LEFT or
//                              HIPBLAS_SIDE_RIGHT (A is on the left or right
//                              side)
//
// hipblasFillMode_t uplo,    // HIPBLAS_FILL_MODE_LOWER (0) or
//                              HIPBLAS_FILL_MODE_UPPER (1), lower or upper part
//                              is referenced
//
// int m, int n              // number of rows (m)  or cols (n) of matrix C and
//                              B, with matrix A sized accordingly.
//
// const double *alpha,      // <type> scalar used for multiplication
//
// const double *A,          // <type> array of dimension lda x m with
//                              lda>=max(1,m) if side == HIPBLAS_SIDE_LEFT and
//                              lda x n with lda>=max(1,n) otherwise.
//
// const double *B,          // <type> array of dimension ldb x n with
//                              ldb>=max(1,m).
//
// const double *beta,       // <type> scalar used for multiplication, if
//                              beta == 0 then C does not have to be a valid
//                              input.
//
// double *C                 // <type> array of dimension ldb x n with
//                              ldb>=max(1,m).
//
// int lda, ldb, ldc         // leading dimension of two-dimensional array used
//                              to store matrix A or B or C
//
// hipblasStatus_t hipblasDsymm(hipblasHandle_t handle,
//                            hipblasSideMode_t side, hipblasFillMode_t uplo,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *B, int ldb,
//                            const double          *beta,
//                            double          *C, int ldc)

//
//
//
// alpha*A(x) + beta*y
// hipblasOperation_t trans,      // operation op(A) that is non- or (conj.)
//                                  transpose. HIPBLAS_OP_N/T/H
// int m, int n,                 // number of rows/cols of A
// const double *x,              // vector x
// double *y,                    // vector y
// int incx, incy                // stride between consecutive elements of x/y.
//
// hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *x, int incx,
//                            const double          *beta,
//                            double          *y, int incy)
