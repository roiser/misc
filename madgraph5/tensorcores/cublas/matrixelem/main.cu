#include "dev_array.h"
#include "hst_matrix.h"
#include "kernel.h"

#include <hip/hip_complex.h>
//#include <hipblas.h>
//#include <hipblas.h>
#include <hipblas.h>

//#define MG5EXAMPLE
#define CUBLAS

/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/

int main() {

#if defined(MG5EXAMPLE)
  const int dim = 24;
  const int M = 2, K = dim, N = dim, SA = M * K, SB = K * N, SC = M * N;
  double _A_mat_[SA], _B_mat_[SB], C_rm[SC];
  dev_array<double> d_A(SA), d_B(SB), d_C(SC);

  fill2(_A_mat_, _B_mat_, C_rm, M, N, K);
  d_A.set(_A_mat_, SA);
  d_B.set(_B_mat_, SB);

  mmult<M, N, K><<<9, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C_rm, SC);
  hipDeviceSynchronize();

  print(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N, K);

#elif defined(CUBLAS)

  // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemv

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasOperation_t trans = HIPBLAS_OP_N;
  int m = 8, n = 8, lda = 0, incx = 0, incy = 0;
  hipDoubleComplex *alpha = 0, *A = 0, *x = 0, *beta = 0, *y = 0;

  // hipblasHandle_t handle,
  // hipblasOperation_t trans,
  // int m, int n,
  // const hipDoubleComplex *alpha,
  // const hipDoubleComplex *A, int lda,
  // const hipDoubleComplex *x, int incx,
  // const hipDoubleComplex *beta,
  // hipDoubleComplex *y, int incy

  // Do the actual multiplication
  hipblasZgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);

  // Destroy the handle
  hipblasDestroy(handle);

#else  // simple example
  const int M = 8, N = 8, K = 4, SA = M * K, SB = K * N, SC = M * N;
  double _A_mat_[SA], _B_mat_[SB], C_rm[SC];
  dev_array<double> d_A(SA), d_B(SB), d_C(SC);

  fill(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N);
  d_A.set(_A_mat_, SA);
  d_B.set(_B_mat_, SB);

  mult<M, N, K><<<1, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C_rm, SC);
  hipDeviceSynchronize();

  print(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N, K);
#endif // MG5EXAMPLE
  return 0;
}
