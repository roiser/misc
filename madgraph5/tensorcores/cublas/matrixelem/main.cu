#include "hip/hip_runtime.h"
//#define DOUBLEPRECISION
//#define TRANSPOSE

#ifdef DOUBLEPRECISION
#define TTYPE double
#define CUB_SYMM hipblasDsymm
#define CUB_GEMV hipblasDgemv
#else
#define TTYPE float
#define CUB_SYMM hipblasSsymm
#define CUB_GEMV hipblasSgemv
#endif

#include "data.h"

#include <iostream>
#include <hipblas.h>

/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/

int mult(hipblasHandle_t handle, const TTYPE *d_A, const TTYPE *d_B, TTYPE *d_C, TTYPE *d_y, TTYPE *h_y, int dsize, const TTYPE* d_Bt = 0) {

  hipblasStatus_t cublas_status;
  hipError_t cuda_status;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = HIPBLAS_OP_N;

  int m = 24, n = 1, lda = 24, ldb = 24, ldc = 24;
  TTYPE alpha = 1, beta = 0;

  cublas_status = CUB_SYMM(handle, side, uplo, m, n, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);

  int incx = 1, incy = 1;
  m = 1;
  n = 24;
  lda = 1;

  if (d_Bt) 
    cublas_status = CUB_GEMV(handle, trans, m, n, &alpha, d_Bt, lda, d_C, incx, &beta, d_y, incy);
  else 
    cublas_status = CUB_GEMV(handle, trans, m, n, &alpha, d_B, lda, d_C, incx, &beta, d_y, incy);
  cuda_status = hipMemcpy(h_y, d_y, dsize, hipMemcpyDeviceToHost);

  return max(cublas_status, cuda_status);;

}

int main() {

  hipblasHandle_t handle;
  hipError_t cuda_status;

  int dsize = sizeof(TTYPE),
      vsize = dsize * medim,
      msize = vsize * medim,
      mult_status = 0;
  const TTYPE
    *h_A = (TTYPE *)malloc(msize),
    *h_B = (TTYPE *)malloc(vsize),
    *d_A, *d_B;
  TTYPE
    *h_C = (TTYPE *)malloc(vsize), 
    *h_y = (TTYPE *)malloc(dsize),
    *d_C, *d_y, me = 0;

  cuda_status = hipMalloc((void**) &d_A, msize);
  cuda_status = hipMalloc((void**) &d_B, vsize);
  cuda_status = hipMalloc((void**) &d_C, vsize);
  cuda_status = hipMalloc((void**) &d_y, dsize);

  memcpy((void*)h_A, &cf[0], msize);
  cuda_status = hipMemcpy((void*)d_A, h_A, msize, hipMemcpyHostToDevice);

  hipblasCreate(&handle);

  memcpy((void*)h_B, &jamp0r[0], vsize);
  cuda_status = hipMemcpy((void*)d_B, h_B, vsize, hipMemcpyHostToDevice);

  mult_status = mult(handle, d_A, d_B, d_C, d_y, h_y, dsize);
  me += *h_y;

  memcpy((void*)h_B, &jamp0i[0], vsize);
  cuda_status = hipMemcpy((void*)d_B, h_B, vsize, hipMemcpyHostToDevice);

  TTYPE *d_Bt = 0;
#ifdef TRANSPOSE
  TTYPE *h_Bt =  (TTYPE *)malloc(vsize),
  for (int i= 0; i < medim; ++i)  h_Bt[i] = -1 * h_B[i];
  cuda_status = hipMalloc((void**) &d_Bt, vsize);
  cuda_status = hipMemcpy((void*)d_Bt, h_Bt, vsize, hipMemcpyHostToDevice);
#endif

  mult_status = mult(handle, d_A, d_B, d_C, d_y, h_y, dsize, d_Bt);
  me += *h_y;

  std::cout << "y: " << me << std::endl;

  hipblasDestroy(handle);

  return max(mult_status, cuda_status);

}


// https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemv

// alpha*A*B + beta*C (side=left) or alpha*B*A + beta*C (side=right),  A is symmetric
// hipblasHandle_t handle,    // 
// hipblasSideMode_t side     // HIPBLAS_SIDE_LEFT or HIPBLAS_SIDE_RIGHT (A is on the left or right side)
// hipblasFillMode_t uplo,    // HIPBLAS_FILL_MODE_LOWER (0) or HIPBLAS_FILL_MODE_UPPER (1), lower or upper part is referenced
// int m, int n,             // number of rows (m) or cols (n) of matrix C and B, with matrix A sized accordingly. 
// const double *alpha,      // <type> scalar used for multiplication
// const double *A,          // <type> array of dimension lda x m with lda>=max(1,m) if side == HIPBLAS_SIDE_LEFT and lda x n with lda>=max(1,n) otherwise.
// const double *B,          // <type> array of dimension ldb x n with ldb>=max(1,m). 
// const double *beta,       // <type> scalar used for multiplication, if beta == 0 then C does not have to be a valid input.
// double *C                 // <type> array of dimension ldb x n with ldb>=max(1,m).
// int lda, ldb, ldc         // leading dimension of two-dimensional array used to store matrix A or B or C

// hipblasStatus_t hipblasDsymm(hipblasHandle_t handle,
//                            hipblasSideMode_t side, hipblasFillMode_t uplo,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *B, int ldb,
//                            const double          *beta,
//                            double          *C, int ldc)



// alpha*A(x) + beta*y
// hipblasOperation_t trans,      // operation op(A) that is non- or (conj.) transpose. HIPBLAS_OP_N/T/H
// int m, int n,                 // number of rows/cols of A
// const double *x,              // vector x
// double *y,                    // vector y
// int incx, incy                // stride between consecutive elements of x/y. 


// hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *x, int incx,
//                            const double          *beta,
//                            double          *y, int incy)

