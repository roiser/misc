#include "hip/hip_runtime.h"
#include "dev_array.h"
#include "hst_matrix.h"
#include "kernel.h"

#include <hip/hip_complex.h>
//#include <hipblas.h>
//#include <hipblas.h>
#include <hipblas.h>

//#define MG5EXAMPLE
#define CUBLAS

/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/

int main() {

#if defined(MG5EXAMPLE)
  const int dim = 24;
  const int M = 2, K = dim, N = dim, SA = M * K, SB = K * N, SC = M * N;
  double _A_mat_[SA], _B_mat_[SB], C_rm[SC];
  dev_array<double> d_A(SA), d_B(SB), d_C(SC);

  fill2(_A_mat_, _B_mat_, C_rm, M, N, K);
  d_A.set(_A_mat_, SA);
  d_B.set(_B_mat_, SB);

  mmult<M, N, K><<<9, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C_rm, SC);
  hipDeviceSynchronize();

  print(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N, K);

#elif defined(CUBLAS)

  // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemv

  // Create a handle for CUBLAS
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
  int m = 24, n = 1, lda = 24, ldb = 1, ldc = 1;
  const double alpha = 1, beta = 0, *A, *B;
  double *C;

// alpha*A*B + beta*C (side=left) or alpha*B*A + beta*C (side=right),  A is symmetric
// hipblasHandle_t handle,    // 
// hipblasSideMode_t side     // HIPBLAS_SIDE_LEFT or HIPBLAS_SIDE_RIGHT (A is on the left or right side) 
// hipblasFillMode_t uplo,    // HIPBLAS_FILL_MODE_LOWER (0) or HIPBLAS_FILL_MODE_UPPER (1), lower or upper part is referenced
// int m, int n,             // number of rows (m) or cols (n) of matrix C and B, with matrix A sized accordingly. 
// const double *alpha,      // <type> scalar used for multiplication
// const double *A,          // <type> array of dimension lda x m with lda>=max(1,m) if side == HIPBLAS_SIDE_LEFT and lda x n with lda>=max(1,n) otherwise.
// const double *B,          // <type> array of dimension ldb x n with ldb>=max(1,m). 
// const double *beta,       // <type> scalar used for multiplication, if beta == 0 then C does not have to be a valid input.
// double *C                 // <type> array of dimension ldb x n with ldb>=max(1,m).
// int lda, ldb, ldc         // leading dimension of two-dimensional array used to store matrix A or B or C

// hipblasStatus_t hipblasDsymm(hipblasHandle_t handle,
//                            hipblasSideMode_t side, hipblasFillMode_t uplo,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *B, int ldb,
//                            const double          *beta,
//                            double          *C, int ldc)

  // matrix mult with sym matrix
  hipblasDsymm(handle, side, uplo, m, n, &alpha, A, lda, B, ldb, &beta, C, ldc);

  // Destroy the handle
  hipblasDestroy(handle);


  // --> old stuff

  // hipblasOperation_t trans = HIPBLAS_OP_N;
  // int m = 8, n = 8, lda = 0, incx = 0, incy = 0;
  // hipDoubleComplex *alpha = 0, *A = 0, *x = 0, *beta = 0, *y = 0;

  // hipblasHandle_t handle,
  // hipblasOperation_t trans,
  // int m, int n,
  // const hipDoubleComplex *alpha,
  // const hipDoubleComplex *A, int lda,
  // const hipDoubleComplex *x, int incx,
  // const hipDoubleComplex *beta,
  // hipDoubleComplex *y, int incy

  // Do the actual multiplication, this was for double complex, not needed !! (and doesn't work)
  // hipblasZgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);


#else  // simple example
  const int M = 8, N = 8, K = 4, SA = M * K, SB = K * N, SC = M * N;
  double _A_mat_[SA], _B_mat_[SB], C_rm[SC];
  dev_array<double> d_A(SA), d_B(SB), d_C(SC);

  fill(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N);
  d_A.set(_A_mat_, SA);
  d_B.set(_B_mat_, SB);

  mult<M, N, K><<<1, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C_rm, SC);
  hipDeviceSynchronize();

  print(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N, K);
#endif // MG5EXAMPLE
  return 0;
}
