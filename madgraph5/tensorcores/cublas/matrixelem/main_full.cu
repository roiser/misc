#include "hip/hip_runtime.h"
#include "dev_array.h"
#include "hst_matrix.h"
#include "kernel.h"

#include <iostream>
#include <hip/hip_complex.h>
//#include <hipblas.h>
//#include <hipblas.h>
#include <hipblas.h>

//#define MG5EXAMPLE
#define CUBLAS

/*
Docu
https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma
https://docs.nvidia.com/deeplearning/performance/dl-performance-matrix-multiplication/index.html
Matrices are (row/column) --> A (M/K), B(K/N), C(M/N)
*/

int main() {

#if defined(MG5EXAMPLE)
  const int dim = 24;
  const int M = 2, K = dim, N = dim, SA = M * K, SB = K * N, SC = M * N;
  double _A_mat_[SA], _B_mat_[SB], C_rm[SC];
  dev_array<double> d_A(SA), d_B(SB), d_C(SC);

  fill2(_A_mat_, _B_mat_, C_rm, M, N, K);
  d_A.set(_A_mat_, SA);
  d_B.set(_B_mat_, SB);

  mmult<M, N, K><<<9, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C_rm, SC);
  hipDeviceSynchronize();

  print(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N, K);

#elif defined(CUBLAS)

  // https://docs.nvidia.com/cuda/cublas/index.html#cublas-lt-t-gt-gemv


  //
  // first multiplication
  //
  hipblasHandle_t handle;
  hipError_t cuda_error;
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  int m = 24, n = 1, lda = 24, ldb = 24, ldc = 24,
//      asize = cfmat_sym * sizeof(double),
      asize = cfmat * sizeof(double),
      bsize = medim * sizeof(double);
  const double alpha = 1, beta = 0,
    *h_A = (double *)malloc(asize),
    *h_B = (double *)malloc(bsize),
    *d_A, *d_B;
  double *h_C = (double *)malloc(bsize), *d_C;
  hipblasStatus_t cublas_error;

  hipblasCreate(&handle);

  // for (int i = 0; i < 24; ++i) {
  //   for (int j = 0; j < 24; ++j) {
  //     if (i != j) cf[i*24 + j] = 0;
  //   }
  // }


  memcpy((void*)h_A, &cf[0], asize);
  memcpy((void*)h_B, &jamp0r[0], bsize);

  cuda_error = hipMalloc((void**) &d_A, asize);
  if (cuda_error) std::cout << "cuda error code: " << cuda_error << std::endl;

  cuda_error = hipMalloc((void**) &d_B, bsize);
  if (cuda_error) std::cout << "cuda error code: " << cuda_error << std::endl;

  cuda_error = hipMalloc((void**) &d_C, bsize);
  if (cuda_error) std::cout << "cuda error code: " << cuda_error << std::endl;

  cuda_error = hipMemcpy((void*)d_A, h_A, asize, hipMemcpyHostToDevice);
  if (cuda_error) std::cout << "cuda error code: " << cuda_error << std::endl;

  cuda_error = hipMemcpy((void*)d_B, h_B, bsize, hipMemcpyHostToDevice);
  if (cuda_error) std::cout << "cuda error code: " << cuda_error << std::endl;

  for (int i = 0; i < 24; ++i) {
    for (int j = 0; j < 24; ++j) {
      std::cout <<  cf[i*24 + j] << " ";
    }
    std::cout << std::endl;
  }


  cublas_error = hipblasDsymm(handle, side, uplo, m, n, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc);
  if (cublas_error) std::cout << "cublas error code: " << cublas_error << std::endl;

  // hipblasStatus_t hipblasDgemv(hipblasHandle_t handle, hipblasOperation_t trans,
  //                            int m, int n,
  //                            const double          *alpha,
  //                            const double          *A, int lda,
  //                            const double          *x, int incx,
  //                            const double          *beta,
  //                            double          *y, int incy)
  // 
  // hipblasOperation_t trans = HIPBLAS_OP_N;
  // int n2 = 24, incx = 1;
  // cublas_error = hipblasDgemv(handle, trans, m, n2, &alpha, d_A, lda, d_B, incx, &beta, d_C, incx);
  // if (cublas_error) std::cout << "cublas error code: " << cublas_error << std::endl;

  cuda_error = hipMemcpy(h_C, d_C, bsize, hipMemcpyDeviceToHost);
  if (cuda_error) std::cout << "cuda error code: " << cuda_error << std::endl;

  hipblasDestroy(handle);

  if (cuda_error) std::cout << "error code: " << cuda_error << std::endl;

  for (int i = 0; i < medim; ++i) std::cout << h_C[i] << std::endl;
  std::cout << std::endl;

  return max(cuda_error, cublas_error);



// alpha*A*B + beta*C (side=left) or alpha*B*A + beta*C (side=right),  A is symmetric
// hipblasHandle_t handle,    // 
// hipblasSideMode_t side     // HIPBLAS_SIDE_LEFT or HIPBLAS_SIDE_RIGHT (A is on the left or right side)
// hipblasFillMode_t uplo,    // HIPBLAS_FILL_MODE_LOWER (0) or HIPBLAS_FILL_MODE_UPPER (1), lower or upper part is referenced
// int m, int n,             // number of rows (m) or cols (n) of matrix C and B, with matrix A sized accordingly. 
// const double *alpha,      // <type> scalar used for multiplication
// const double *A,          // <type> array of dimension lda x m with lda>=max(1,m) if side == HIPBLAS_SIDE_LEFT and lda x n with lda>=max(1,n) otherwise.
// const double *B,          // <type> array of dimension ldb x n with ldb>=max(1,m). 
// const double *beta,       // <type> scalar used for multiplication, if beta == 0 then C does not have to be a valid input.
// double *C                 // <type> array of dimension ldb x n with ldb>=max(1,m).
// int lda, ldb, ldc         // leading dimension of two-dimensional array used to store matrix A or B or C

// hipblasStatus_t hipblasDsymm(hipblasHandle_t handle,
//                            hipblasSideMode_t side, hipblasFillMode_t uplo,
//                            int m, int n,
//                            const double          *alpha,
//                            const double          *A, int lda,
//                            const double          *B, int ldb,
//                            const double          *beta,
//                            double          *C, int ldc)



  // --> old stuff

  // hipblasOperation_t trans = HIPBLAS_OP_N;
  // int m = 8, n = 8, lda = 0, incx = 0, incy = 0;
  // hipDoubleComplex *alpha = 0, *A = 0, *x = 0, *beta = 0, *y = 0;

  // hipblasHandle_t handle,
  // hipblasOperation_t trans,
  // int m, int n,
  // const hipDoubleComplex *alpha,
  // const hipDoubleComplex *A, int lda,
  // const hipDoubleComplex *x, int incx,
  // const hipDoubleComplex *beta,
  // hipDoubleComplex *y, int incy

  // Do the actual multiplication, this was for double complex, not needed !! (and doesn't work)
  // hipblasZgemv(handle, trans, m, n, alpha, A, lda, x, incx, beta, y, incy);


#else  // simple example
  const int M = 8, N = 8, K = 4, SA = M * K, SB = K * N, SC = M * N;
  double _A_mat_[SA], _B_mat_[SB], C_rm[SC];
  dev_array<double> d_A(SA), d_B(SB), d_C(SC);

  fill(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N);
  d_A.set(_A_mat_, SA);
  d_B.set(_B_mat_, SB);

  mult<M, N, K><<<1, 32>>>(d_A.getData(), d_B.getData(), d_C.getData());
  hipDeviceSynchronize();
  d_C.get(C_rm, SC);
  hipDeviceSynchronize();

  print(_A_mat_, _B_mat_, C_rm, _A_rdm_, _A_cdm_, _B_rdm_, _B_cdm_, M, N, K);
#endif // MG5EXAMPLE
  return 0;
}
